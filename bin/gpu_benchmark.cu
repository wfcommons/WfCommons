#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cstdlib>  // For std::atoi
#include "gpu_benchmark.h"

// Kernel function to perform a simple workload
__global__ void simpleKernel(int* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] * data[idx];  // Simple workload: squaring each element
    }
}

// Function to run the GPU benchmark with no time limit
void runBenchmark(int max_work) {
    int* h_data = new int[max_work];
    int* d_data;

    // Initialize data
    for (int i = 0; i < max_work; i++) {
        h_data[i] = i;
    }

    // Allocate GPU memory
    hipMalloc(&d_data, max_work * sizeof(int));

    // Copy data to GPU
    hipMemcpy(d_data, h_data, max_work * sizeof(int), hipMemcpyHostToDevice);

    // Kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (max_work + threadsPerBlock - 1) / threadsPerBlock;

    // Run the kernel
    simpleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, max_work);

    // Ensure the kernel has finished executing
    hipDeviceSynchronize();

    // Copy results back to host (optional, just for validation)
    hipMemcpy(h_data, d_data, max_work * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_data);
    delete[] h_data;

    std::cout << "Benchmark completed!" << std::endl;
}

// Function to run the GPU benchmark for a specified time
void runBenchmarkTime(int max_work, int runtime_in_seconds) {
    int* h_data = new int[max_work];
    int* d_data;

    // Initialize data
    for (int i = 0; i < max_work; i++) {
        h_data[i] = i;
    }

    // Allocate GPU memory
    hipMalloc(&d_data, max_work * sizeof(int));

    // Copy data to GPU
    hipMemcpy(d_data, h_data, max_work * sizeof(int), hipMemcpyHostToDevice);

    // Start the timer
    auto start = std::chrono::high_resolution_clock::now();

    // Kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (max_work + threadsPerBlock - 1) / threadsPerBlock;

    // Run the workload loop until the specified runtime is reached
    while (std::chrono::duration_cast<std::chrono::seconds>(std::chrono::high_resolution_clock::now() - start).count() < runtime_in_seconds) {
        simpleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, max_work);
        hipDeviceSynchronize();  // Ensure the kernel has finished executing
    }

    // Copy results back to host (optional, just for validation)
    hipMemcpy(h_data, d_data, max_work * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_data);
    delete[] h_data;

    std::cout << "Benchmark completed!" << std::endl;
}

int main(int argc, char* argv[]) {
    // Check for the correct number of command line arguments
    if (argc == 2) {
        // Parse the command line arguments
        int max_work = std::atoi(argv[1]);

        // Validate the input arguments
        if (max_work <= 0) {
            std::cerr << "max_work must be a positive integer." << std::endl;
            return 1;
        }

        runBenchmark(max_work);

    } else if (argc == 3) {
        // Parse the command line arguments
        int max_work = std::atoi(argv[1]);
        int runtime_in_seconds = std::atoi(argv[2]);

        // Validate the input arguments
        if (max_work <= 0 || runtime_in_seconds <= 0) {
            std::cerr << "Both max_work and runtime_in_seconds must be positive integers." << std::endl;
            return 1;
        }

        runBenchmarkTime(max_work, runtime_in_seconds);

    } else {
        std::cerr << "Usage: " << argv[0] << " <max_work> [runtime_in_seconds]" << std::endl;
        return 1;
    }

    return 0;
}
