#include "hip/hip_runtime.h"
#include "kernels.cuh"




__global__ void setup_kernel(hiprandState *state)
{
	int index = threadIdx.x + blockDim.x*blockIdx.x;
    hiprand_init(123456789, index, 0, &state[index]);
}




__global__ void monte_carlo_kernel(hiprandState *state, int *count, int m)
{
	unsigned int index_x = threadIdx.x + blockDim.x*blockIdx.x;
	unsigned int index_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	__shared__ int cache[16*16];
	// cache[threadIdx.x] = 0;
	// cache[threadIdx.y] = 0;

	// __syncthreads();

	
	unsigned int temp = 0;
	while(temp < m){
		unsigned u = threadIdx.y*blockDim.x + threadIdx.x;
		unsigned i = hiprand_uniform(&state[index_x]);
		unsigned j = hiprand_uniform(&state[index_y]);
		unsigned Ni = gridDim.x*blockDim.x;
		unsigned Nj = gridDim.y*blockDim.y;

		float x = i/(float)Ni;
		float y = j/(float)Nj;
		float r = std::sqrt(x*x + y*y);

		cache[u] += r<=1; 
		temp++; 
	}

	
	// reduction --- probably remove this part 
	// int i = blockDim.x/2;
	// while(i != 0){
	// 	if(threadIdx.x < i){
	// 		cache[threadIdx.x] += cache[threadIdx.x + i];
	// 	}

	// 	i /= 2;
	// 	__syncthreads();
	// }


	// // update to our global variable count
	// if(threadIdx.x == 0){
	// 	atomicAdd(count, cache[0]);
	// }
}



